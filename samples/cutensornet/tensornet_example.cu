#include "hip/hip_runtime.h"
/*  
 * Copyright (c) 2021-2022, NVIDIA CORPORATION & AFFILIATES.
 * 
 * SPDX-License-Identifier: BSD-3-Clause
 */  

// Sphinx: #1
#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>
#include <cassert>

#include <hip/hip_runtime.h>
#include <cutensornet.h>
#include <hiptensor.h>

#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
if( err != CUTENSORNET_STATUS_SUCCESS )                                \
{ printf("Error: %s in line %d\n", cutensornetGetErrorString(err), __LINE__); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{  const auto err = x;                                             \
   if( err != hipSuccess )                                        \
   { printf("Error: %s in line %d\n", hipGetErrorString(err), __LINE__); return err; } \
};

struct GPUTimer
{
   GPUTimer(hipStream_t stream): stream_(stream)
   {
      hipEventCreate(&start_);
      hipEventCreate(&stop_);
   }

   ~GPUTimer()
   {
      hipEventDestroy(start_);
      hipEventDestroy(stop_);
   }

   void start()
   {
      hipEventRecord(start_, stream_);
   }

   float seconds()
   {
      hipEventRecord(stop_, stream_);
      hipEventSynchronize(stop_);
      float time;
      hipEventElapsedTime(&time, start_, stop_);
      return time * 1e-3;
   }

   private:
   hipEvent_t start_, stop_;
   hipStream_t stream_;
};


int main()
{
   const size_t cuTensornetVersion = cutensornetGetVersion();
   printf("cuTensorNet-vers:%ld\n",cuTensornetVersion);

   hipDeviceProp_t prop;
   int32_t deviceId = -1;
   HANDLE_CUDA_ERROR( hipGetDevice(&deviceId) );
   HANDLE_CUDA_ERROR( hipGetDeviceProperties(&prop, deviceId) );

   printf("===== device info ======\n");
   printf("GPU-name:%s\n", prop.name);
   printf("GPU-clock:%d\n", prop.clockRate);
   printf("GPU-memoryClock:%d\n", prop.memoryClockRate);
   printf("GPU-nSM:%d\n", prop.multiProcessorCount);
   printf("GPU-major:%d\n", prop.major);
   printf("GPU-minor:%d\n", prop.minor);
   printf("========================\n");

   typedef float floatType;
   hipDataType typeData = HIP_R_32F;
   cutensornetComputeType_t typeCompute = CUTENSORNET_COMPUTE_32F;

   printf("Include headers and define data types\n");

   // Sphinx: #2
   /**********************
   * Computing: D_{m,x,n,y} = A_{m,h,k,n} B_{u,k,h} C_{x,u,y}
   **********************/

   constexpr int32_t numInputs = 3;

   // Create vector of modes
   std::vector<int32_t> modesA{'m','h','k','n'};
   std::vector<int32_t> modesB{'u','k','h'};
   std::vector<int32_t> modesC{'x','u','y'};
   std::vector<int32_t> modesD{'m','x','n','y'};

   // Extents
   std::unordered_map<int32_t, int64_t> extent;
   extent['m'] = 96;
   extent['n'] = 96;
   extent['u'] = 96;
   extent['h'] = 64;
   extent['k'] = 64;
   extent['x'] = 64;
   extent['y'] = 64;

   // Create a vector of extents for each tensor
   std::vector<int64_t> extentA;
   for (auto mode : modesA)
      extentA.push_back(extent[mode]);
   std::vector<int64_t> extentB;
   for (auto mode : modesB)
      extentB.push_back(extent[mode]);
   std::vector<int64_t> extentC;
   for (auto mode : modesC)
      extentC.push_back(extent[mode]);
   std::vector<int64_t> extentD;
   for (auto mode : modesD)
      extentD.push_back(extent[mode]);

   printf("Define network, modes, and extents\n");

   // Sphinx: #3
   /**********************
   * Allocating data
   **********************/

   size_t elementsA = 1;
   for (auto mode : modesA)
      elementsA *= extent[mode];
   size_t elementsB = 1;
   for (auto mode : modesB)
      elementsB *= extent[mode];
   size_t elementsC = 1;
   for (auto mode : modesC)
      elementsC *= extent[mode];
   size_t elementsD = 1;
   for (auto mode : modesD)
      elementsD *= extent[mode];

   size_t sizeA = sizeof(floatType) * elementsA;
   size_t sizeB = sizeof(floatType) * elementsB;
   size_t sizeC = sizeof(floatType) * elementsC;
   size_t sizeD = sizeof(floatType) * elementsD;
   printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC + sizeD)/1024./1024./1024);

   void* rawDataIn_d[numInputs];
   void* D_d;
   HANDLE_CUDA_ERROR(hipMalloc((void**) &rawDataIn_d[0], sizeA));
   HANDLE_CUDA_ERROR(hipMalloc((void**) &rawDataIn_d[1], sizeB));
   HANDLE_CUDA_ERROR(hipMalloc((void**) &rawDataIn_d[2], sizeC));
   HANDLE_CUDA_ERROR(hipMalloc((void**) &D_d, sizeD));

   floatType *A = (floatType*) malloc(sizeof(floatType) * elementsA);
   floatType *B = (floatType*) malloc(sizeof(floatType) * elementsB);
   floatType *C = (floatType*) malloc(sizeof(floatType) * elementsC);
   floatType *D = (floatType*) malloc(sizeof(floatType) * elementsD);

   if (A == NULL || B == NULL || C == NULL || D == NULL)
   {
      printf("Error: Host allocation of A or C.\n");
      return -1;

   }
   /**********************
   * Allocate workspace
   **********************/

   size_t freeMem, totalMem;
   HANDLE_CUDA_ERROR( hipMemGetInfo(&freeMem, &totalMem ));

   uint64_t worksize = freeMem * 0.9;

   void *work = nullptr;
   HANDLE_CUDA_ERROR( hipMalloc(&work, worksize) );

   /*******************
   * Initialize data
   *******************/

   for (uint64_t i = 0; i < elementsA; i++)
      A[i] = ((float) rand())/RAND_MAX;
   for (uint64_t i = 0; i < elementsB; i++)
      B[i] = ((float) rand())/RAND_MAX;
   for (uint64_t i = 0; i < elementsC; i++)
      C[i] = ((float) rand())/RAND_MAX;
   memset(D, 0, sizeof(floatType) * elementsD);

   HANDLE_CUDA_ERROR(hipMemcpy(rawDataIn_d[0], A, sizeA, hipMemcpyHostToDevice));
   HANDLE_CUDA_ERROR(hipMemcpy(rawDataIn_d[1], B, sizeB, hipMemcpyHostToDevice));
   HANDLE_CUDA_ERROR(hipMemcpy(rawDataIn_d[2], C, sizeC, hipMemcpyHostToDevice));

   printf("Allocate memory for data and workspace, and initialize data.\n");

   // Sphinx: #4
   /*************************
   * cuTensorNet
   *************************/

   hipStream_t stream;
   hipStreamCreate(&stream);

   cutensornetHandle_t handle;
   HANDLE_ERROR(cutensornetCreate(&handle));

   const int32_t nmodeA = modesA.size();
   const int32_t nmodeB = modesB.size();
   const int32_t nmodeC = modesC.size();
   const int32_t nmodeD = modesD.size();

   /*******************************
   * Create Network Descriptor
   *******************************/

   const int32_t* modesIn[] = {modesA.data(), modesB.data(), modesC.data()};
   int32_t const numModesIn[] = {nmodeA, nmodeB, nmodeC};
   const int64_t* extentsIn[] = {extentA.data(), extentB.data(), extentC.data()};
   const int64_t* stridesIn[] = {NULL, NULL, NULL}; // strides are optional; if no stride is provided, then cuTensorNet assumes a generalized column-major data layout

   // Notice that pointers are allocated via hipMalloc are aligned to 256 byte
   // boundaries by default; however here we're checking the pointer alignment explicitly
   // to demonstrate how one would check the alginment for arbitrary pointers.

   auto getMaximalPointerAlignment = [](const void* ptr) {
      const uint64_t ptrAddr  = reinterpret_cast<uint64_t>(ptr);
      uint32_t alignment = 1;
      while(ptrAddr % alignment == 0 &&
            alignment < 256) // at the latest we terminate once the alignment reached 256 bytes (we could be going, but any alignment larger or equal to 256 is equally fine)
      {
         alignment *= 2;
      }
      return alignment;
   };
   const uint32_t alignmentsIn[] = {getMaximalPointerAlignment(rawDataIn_d[0]),
                                    getMaximalPointerAlignment(rawDataIn_d[1]),
                                    getMaximalPointerAlignment(rawDataIn_d[2])};
   const uint32_t alignmentOut = getMaximalPointerAlignment(D_d);

   // setup tensor network
   cutensornetNetworkDescriptor_t descNet;
   HANDLE_ERROR (cutensornetCreateNetworkDescriptor(handle,
                                                numInputs, numModesIn, extentsIn, stridesIn, modesIn, alignmentsIn,
                                                nmodeD, extentD.data(), /*stridesOut = */NULL, modesD.data(), alignmentOut,
                                                typeData, typeCompute,
                                                &descNet));

   printf("Initialize the cuTensorNet library and create a network descriptor.\n");

   // Sphinx: #5
   /*******************************
   * Find "optimal" contraction order and slicing
   *******************************/

   cutensornetContractionOptimizerConfig_t optimizerConfig;
   HANDLE_ERROR (cutensornetCreateContractionOptimizerConfig(handle, &optimizerConfig));

   // Set the value of the partitioner imbalance factor, if desired
   int imbalance_factor = 30;
   HANDLE_ERROR(cutensornetContractionOptimizerConfigSetAttribute(
                                                               handle,
                                                               optimizerConfig,
                                                               CUTENSORNET_CONTRACTION_OPTIMIZER_CONFIG_GRAPH_IMBALANCE_FACTOR,
                                                               &imbalance_factor,
                                                               sizeof(imbalance_factor)));

   cutensornetContractionOptimizerInfo_t optimizerInfo;
   HANDLE_ERROR (cutensornetCreateContractionOptimizerInfo(handle, descNet, &optimizerInfo));

   HANDLE_ERROR (cutensornetContractionOptimize(handle,
                                             descNet,
                                             optimizerConfig,
                                             worksize,
                                             optimizerInfo));

   int64_t numSlices = 0;
   HANDLE_ERROR( cutensornetContractionOptimizerInfoGetAttribute(
               handle,
               optimizerInfo,
               CUTENSORNET_CONTRACTION_OPTIMIZER_INFO_NUM_SLICES,
               &numSlices,
               sizeof(numSlices)));

   assert(numSlices > 0);

   printf("Find an optimized contraction path with cuTensorNet optimizer.\n");

   // Sphinx: #6
   /*******************************
   * Initialize all pair-wise contraction plans (for cuTENSOR)
   *******************************/
   cutensornetContractionPlan_t plan;

   cutensornetWorkspaceDescriptor_t workDesc;
   HANDLE_ERROR(cutensornetCreateWorkspaceDescriptor(handle, &workDesc));

   uint64_t requiredWorkspaceSize = 0;
   HANDLE_ERROR(cutensornetWorkspaceComputeSizes(handle,
                                          descNet,
                                          optimizerInfo,
                                          workDesc));

   HANDLE_ERROR(cutensornetWorkspaceGetSize(handle,
                                         workDesc,
                                         CUTENSORNET_WORKSIZE_PREF_MIN,
                                         CUTENSORNET_MEMSPACE_DEVICE,
                                         &requiredWorkspaceSize));
   if (worksize < requiredWorkspaceSize)
   {
      printf("Not enough workspace memory is available.");
   }
   else
   {
      HANDLE_ERROR (cutensornetWorkspaceSet(handle,
                                            workDesc,
                                            CUTENSORNET_MEMSPACE_DEVICE,
                                            work,
                                            worksize));
                                          
      HANDLE_ERROR( cutensornetCreateContractionPlan(handle,
                                                     descNet,
                                                     optimizerInfo,
                                                     workDesc,
                                                     &plan) );

      /*******************************
      * Optional: Auto-tune cuTENSOR's cutensorContractionPlan to pick the fastest kernel
      *******************************/
      cutensornetContractionAutotunePreference_t autotunePref;
      HANDLE_ERROR(cutensornetCreateContractionAutotunePreference(handle,
                              &autotunePref));

      const int numAutotuningIterations = 5; // may be 0
      HANDLE_ERROR(cutensornetContractionAutotunePreferenceSetAttribute(
                              handle,
                              autotunePref,
                              CUTENSORNET_CONTRACTION_AUTOTUNE_MAX_ITERATIONS,
                              &numAutotuningIterations,
                              sizeof(numAutotuningIterations)));

      // modify the plan again to find the best pair-wise contractions
      HANDLE_ERROR(cutensornetContractionAutotune(handle,
                              plan,
                              rawDataIn_d,
                              D_d,
                              workDesc,
                              autotunePref,
                              stream));

      HANDLE_ERROR(cutensornetDestroyContractionAutotunePreference(autotunePref));

      printf("Create a contraction plan for cuTENSOR and optionally auto-tune it.\n");

      // Sphinx: #7
      /**********************
      * Run
      **********************/
      GPUTimer timer{stream};
      double minTimeCUTENSOR = 1e100;
      const int numRuns = 3; // to get stable perf results
      for (int i=0; i < numRuns; ++i)
      {
         hipMemcpy(D_d, D, sizeD, hipMemcpyHostToDevice); // restore output
         hipDeviceSynchronize();

         /*
         * Contract over all slices.
         *
         * A user may choose to parallelize this loop across multiple devices.
         * (Note, however, that as of cuTensorNet v1.0.0 the contraction must
         * start from slice 0, see the cutensornetContraction documentation at
         * https://docs.nvidia.com/cuda/cuquantum/cutensornet/api/functions.html#cutensornetcontraction )
         */
         for(int64_t sliceId=0; sliceId < numSlices; ++sliceId)
         {
            timer.start();

            HANDLE_ERROR(cutensornetContraction(handle,
                                    plan,
                                    rawDataIn_d,
                                    D_d,
                                    workDesc, sliceId, stream));

            // Synchronize and measure timing
            auto time = timer.seconds();
            minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
         }
      }

      printf("Contract the network, each slice uses the same contraction plan.\n");

      /*************************/

      double flops = -1;

      HANDLE_ERROR( cutensornetContractionOptimizerInfoGetAttribute(
                  handle,
                  optimizerInfo,
                  CUTENSORNET_CONTRACTION_OPTIMIZER_INFO_FLOP_COUNT,
                  &flops,
                  sizeof(flops)));

      printf("numSlices: %ld\n", numSlices);
      printf("%.2f ms / slice\n", minTimeCUTENSOR * 1000.f);
      printf("%.2f GFLOPS/s\n", flops/1e9/minTimeCUTENSOR );
   }

   HANDLE_ERROR(cutensornetDestroy(handle));
   HANDLE_ERROR(cutensornetDestroyNetworkDescriptor(descNet));
   HANDLE_ERROR(cutensornetDestroyContractionPlan(plan));
   HANDLE_ERROR(cutensornetDestroyContractionOptimizerConfig(optimizerConfig));
   HANDLE_ERROR(cutensornetDestroyContractionOptimizerInfo(optimizerInfo));
   HANDLE_ERROR(cutensornetDestroyWorkspaceDescriptor(workDesc));

   if (A) free(A);
   if (B) free(B);
   if (C) free(C);
   if (D) free(D);
   if (rawDataIn_d[0]) hipFree(rawDataIn_d[0]);
   if (rawDataIn_d[1]) hipFree(rawDataIn_d[1]);
   if (rawDataIn_d[2]) hipFree(rawDataIn_d[2]);
   if (D_d) hipFree(D_d);
   if (work) hipFree(work);

   printf("Free resource and exit.\n");

   return 0;
}
