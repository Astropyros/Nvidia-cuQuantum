#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES.  All rights reserved.
 *
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <string.h>           // strcpy
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR

#define SUPPORTS_MEMORY_POOL ( __CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 2))
#if SUPPORTS_MEMORY_POOL

// upon success, this function should return 0, otherwise a nonzero value
int myMemPoolAlloc(void* ctx, void** ptr, size_t size, hipStream_t stream) {
    hipMemPool_t& pool = *static_cast<hipMemPool_t*>(ctx);
    hipError_t status = hipMallocFromPoolAsync(ptr, size, pool, stream);
    return (int)status;
}

// upon success, this function should return 0, otherwise a nonzero value
int myMemPoolFree(void*, void* ptr, size_t, hipStream_t stream) {
    hipError_t status = hipFreeAsync(ptr, stream);
    return (int)status;
}

int main(void) {
    // state vector
    const int nIndexBits   = 3;
    const int nSvSize      = (1 << nIndexBits);

    hipDoubleComplex h_sv[] = {{ 0.48, 0.0}, { 0.36, 0.0}, { 0.64, 0.0}, { 0.48, 0.0}, 
                              { 0.0,  0.0}, { 0.0,  0.0}, { 0.0,  0.0}, { 0.0,  0.0}};

    //----------------------------------------------------------------------------------------------
    // gates
    const int adjoint = 0;
    const custatevecMatrixLayout_t layout = CUSTATEVEC_MATRIX_LAYOUT_ROW;

    // Hadamard gate
    const int hTargets[] = {2};
    const uint32_t hNTargets = 1;
    const double Rsqrt2 = 1. / std::sqrt(2.);
    hipDoubleComplex hGate[] = {{Rsqrt2, 0.0}, {Rsqrt2, 0.0},
                               {Rsqrt2, 0.0}, {-Rsqrt2, 0.0}};

    // control-SWAP gate
    const int swapTargets[] = {0, 1};
    const uint32_t swapNTargets = 2;
    const int swapControls[] = {2};
    const uint32_t swapNControls = 1;
    hipDoubleComplex swapGate[] = {{1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {1.0, 0.0}, {0.0, 0.0}, {0.0, 0.0},
                                  {0.0, 0.0}, {0.0, 0.0}, {0.0, 0.0}, {1.0, 0.0}};

    // observable
    const int basisBits[] = {2};
    const uint32_t nBasisBits = 1;
    hipDoubleComplex observable[] = {{1.0, 0.0}, {0.0, 0.0},
                                    {0.0, 0.0}, {0.0, 0.0}};

    //----------------------------------------------------------------------------------------------
    // device configuration
    int deviceId;
    HANDLE_CUDA_ERROR( hipGetDevice(&deviceId) );

    hipError_t status;
    int isMemPoolSupported;
    status = hipDeviceGetAttribute(&isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, deviceId);
    if (status != hipSuccess || !isMemPoolSupported) {
        printf("memory handler example WAIVED: CUDA Memory pools is not supported.\n");
        return EXIT_SUCCESS;
    }

    hipMemPool_t memPool;
    HANDLE_CUDA_ERROR( hipDeviceGetDefaultMemPool(&memPool, deviceId) );

    // avoid shrinking the pool 
    uint64_t threshold = UINT64_MAX;
    hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &threshold);

    hipStream_t stream;
    HANDLE_CUDA_ERROR( hipStreamCreate(&stream) );

    //----------------------------------------------------------------------------------------------
    // data transfer of state vector
    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR( hipMallocAsync((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex), stream) );

    HANDLE_CUDA_ERROR( hipMemcpyAsync(d_sv, h_sv, nSvSize * sizeof(hipDoubleComplex), 
                                       hipMemcpyHostToDevice, stream) );

    //----------------------------------------------------------------------------------------------
    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );
    HANDLE_ERROR( custatevecSetStream(handle, stream) );

    // device memory handler
    custatevecDeviceMemHandler_t handler;
    handler.ctx = &memPool;
    handler.device_alloc = myMemPoolAlloc;
    handler.device_free = myMemPoolFree;
    strcpy(handler.name, "mempool");
    HANDLE_ERROR( custatevecSetDeviceMemHandler(handle, &handler) );

    // apply Hadamard gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, hGate, HIP_C_64F,
                  layout, adjoint, hTargets, hNTargets, nullptr, nullptr, 0, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // apply control-SWAP gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, swapGate, HIP_C_64F,
                  layout, adjoint, swapTargets, swapNTargets, swapControls, nullptr, swapNControls, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // apply Hadamard gate
    HANDLE_ERROR( custatevecApplyMatrix(
                  handle, d_sv, HIP_C_64F, nIndexBits, hGate, HIP_C_64F,
                  layout, adjoint, hTargets, hNTargets, nullptr, nullptr, 0, 
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    // compute expectation
    double expectationValue;
    HANDLE_ERROR( custatevecComputeExpectation(
                  handle, d_sv, HIP_C_64F, nIndexBits, &expectationValue, HIP_R_64F, nullptr,
                  observable, HIP_C_64F, layout, basisBits, nBasisBits,
                  CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0) );

    HANDLE_CUDA_ERROR( hipStreamSynchronize(stream) );

    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------

    // release device memory and stream
    HANDLE_CUDA_ERROR( hipFreeAsync(d_sv, stream) );
    HANDLE_CUDA_ERROR( hipStreamDestroy(stream) );

    double expectationValueResult = 0.9608;
    bool correct = almost_equal(expectationValue, expectationValueResult);
    if (correct) {
        printf("memory_handler example PASSED\n");
        return EXIT_SUCCESS;
    }
    else {
        printf("memory_handler example FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
}

#else
int main(void) {
    printf("memory_handler example WAIVED : This example uses CUDA's built-in stream-ordered memory allocator, which requires CUDA 11.2+.\n");
    return EXIT_SUCCESS;
}
#endif